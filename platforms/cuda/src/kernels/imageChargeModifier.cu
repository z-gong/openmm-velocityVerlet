#include "hip/hip_runtime.h"
/**
 * Perform the first step of Langevin integration.
 */

extern "C" __global__ void updateImagePositions(real4 *__restrict__ posq,
                                                real4 *__restrict__ posqCorrection,
                                                const int2 *__restrict__ imagePairs,
                                                mixed mirror) {

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < NUM_IMAGES; i += blockDim.x * gridDim.x) {
        int2 pair = imagePairs[i];
        int index_img = pair.x;
        int index_par = pair.y;
//        if (i==0)
//            printf("Mirror = %f; Pair = %d %d, Index= %d %d, z = %f %f\n",
//                    mirror, pair.x, pair.y, index_img, index_par, posq[index_img].z, posq[index_par].z);
        posq[index_img].x = posq[index_par].x;
        posq[index_img].y = posq[index_par].y;
        posqCorrection[index_img].x = posqCorrection[index_par].x;
        posqCorrection[index_img].y = posqCorrection[index_par].y;

#ifdef USE_MIXED_PRECISION
        mixed z = posq[index_par].z + (mixed) posqCorrection[index_par].z;
        z = mirror * 2 - z;
        posq[index_img].z = (real) z;
        posqCorrection[index_img].z = (real) (z - (real) z);
#else
        posq[index_img].z = 2 * mirror - pos.z;
#endif
    }
}
