#include "hip/hip_runtime.h"

/**
 * Perform the velocity update of half step velocity verlet.
 */

extern "C" __global__ void velocityVerletIntegrateVelocities(mixed4 *__restrict__ velm,
                                                             const long long *__restrict__ force,
                                                             const real3 *__restrict__ forceLD,
                                                             mixed4 *__restrict__ posDelta,
                                                             const int *__restrict__ particlesNH,
                                                             const mixed2 *__restrict__ dt,
                                                             const mixed fscale,
                                                             bool updatePosDelta) {

    mixed stepSize = dt[0].y;

    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < NUM_ATOMS; index += blockDim.x * gridDim.x) {
        mixed4 velocity = velm[index];

//        if (abs(forceLD[index].x > 0.00001) || abs(forceLD[index].y > 0.00001) || abs(forceLD[index].z > 0.00001) )
//            printf("FORCELD index %d , %f %f %f\n", index, forceLD[index].x, forceLD[index].y, forceLD[index].z);

        if (velocity.w != 0) {
            velocity.x += 0.5 * stepSize * velocity.w * forceLD[index].x + fscale * velocity.w * force[index];
            velocity.y += 0.5 * stepSize * velocity.w * forceLD[index].y + fscale * velocity.w * force[index + PADDED_NUM_ATOMS];
            velocity.z += 0.5 * stepSize * velocity.w * forceLD[index].z + fscale * velocity.w * force[index + PADDED_NUM_ATOMS * 2];
            velm[index] = velocity;
            if (updatePosDelta) {
                posDelta[index] = make_mixed4(stepSize * velocity.x, stepSize * velocity.y, stepSize * velocity.z, 0);
            }
        }
    }
}

/**
 * Perform the position update.
 */

extern "C" __global__ void velocityVerletIntegratePositions(real4 *__restrict__ posq,
                                                            real4 *__restrict__ posqCorrection,
                                                            const mixed4 *__restrict__ posDelta,
                                                            mixed4 *__restrict__ velm,
                                                            const mixed2 *__restrict__ dt,
                                                            const int *__restrict__ particlesNH) {
    double invStepSize = 1.0 / dt[0].y;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < NUM_ATOMS; index += blockDim.x * gridDim.x) {
        mixed4 vel = velm[index];
        if (vel.w != 0) {
#ifdef USE_MIXED_PRECISION

            real4 pos1 = posq[index];
            real4 pos2 = posqCorrection[index];
            mixed4 pos = make_mixed4(pos1.x+(mixed)pos2.x, pos1.y+(mixed)pos2.y, pos1.z+(mixed)pos2.z, pos1.w);
#else
            real4 pos = posq[index];
#endif
            mixed4 delta = posDelta[index];
            pos.x += delta.x;
            pos.y += delta.y;
            pos.z += delta.z;
            vel.x = (mixed) (invStepSize*delta.x);
            vel.y = (mixed) (invStepSize*delta.y);
            vel.z = (mixed) (invStepSize*delta.z);
#ifdef USE_MIXED_PRECISION
            posq[index] = make_real4((real) pos.x, (real) pos.y, (real) pos.z, (real) pos.w);
            posqCorrection[index] = make_real4(pos.x-(real) pos.x, pos.y-(real) pos.y, pos.z-(real) pos.z, 0);
#else
            posq[index] = pos;
#endif
            velm[index] = vel;
        }
    }
}

/**
 * Apply hard wall constraints
 */
extern "C" __global__ void applyHardWallConstraints(real4 *__restrict__ posq,
                                                    real4 *__restrict__ posqCorrection,
                                                    mixed4 *__restrict__ velm,
                                                    const int2 *__restrict__ allPairs,
                                                    const mixed2 *__restrict__ dt,
                                                    mixed maxDrudeDistance,
                                                    mixed hardwallscaleDrude) {

    mixed stepSize = dt[0].y;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_ALL_PAIRS; i += blockDim.x*gridDim.x) {
        int2 particles = allPairs[i];
#ifdef USE_MIXED_PRECISION
        real4 posReal1 = posq[particles.x];
        real4 posReal2 = posq[particles.y];
        real4 posCorr1 = posqCorrection[particles.x];
        real4 posCorr2 = posqCorrection[particles.y];
        mixed4 pos1 = make_mixed4(posReal1.x+(mixed)posCorr1.x, posReal1.y+(mixed)posCorr1.y, posReal1.z+(mixed)posCorr1.z, posReal1.w);
        mixed4 pos2 = make_mixed4(posReal2.x+(mixed)posCorr2.x, posReal2.y+(mixed)posCorr2.y, posReal2.z+(mixed)posCorr2.z, posReal2.w);
#else
        mixed4 pos1 = posq[particles.x];
        mixed4 pos2 = posq[particles.y];
#endif
        mixed4 delta = pos1-pos2;
        mixed r = SQRT(delta.x*delta.x + delta.y*delta.y + delta.z*delta.z);
        mixed rInv = RECIP(r);
        if (rInv*maxDrudeDistance < 1) {
            // The constraint has been violated, so make the inter-particle distance "bounce"
            // off the hard wall.

            mixed4 bondDir = delta*rInv;
            mixed4 vel1 = velm[particles.x];
            mixed4 vel2 = velm[particles.y];
            mixed mass1 = RECIP(vel1.w);
            mixed mass2 = RECIP(vel2.w);
            mixed deltaR = r-maxDrudeDistance;
            mixed deltaT = stepSize;
            mixed dotvr1 = vel1.x*bondDir.x + vel1.y*bondDir.y + vel1.z*bondDir.z;
            mixed4 vb1 = bondDir*dotvr1;
            mixed4 vp1 = vel1-vb1;
            if (vel2.w == 0) {
                // The parent particle is massless, so move only the Drude particle.

                if (dotvr1 != 0)
                    deltaT = deltaR/fabs(dotvr1);
                if (deltaT > stepSize)
                    deltaT = stepSize;
                dotvr1 = -dotvr1*hardwallscaleDrude/(fabs(dotvr1)*SQRT(mass1));
                mixed dr = -deltaR + deltaT*dotvr1;
                pos1.x += bondDir.x*dr;
                pos1.y += bondDir.y*dr;
                pos1.z += bondDir.z*dr;
#ifdef USE_MIXED_PRECISION
                posq[particles.x] = make_real4((real) pos1.x, (real) pos1.y, (real) pos1.z, (real) pos1.w);
                posqCorrection[particles.x] = make_real4(pos1.x-(real) pos1.x, pos1.y-(real) pos1.y, pos1.z-(real) pos1.z, 0);
#else
                posq[particles.x] = pos1;
#endif
                vel1.x = vp1.x + bondDir.x*dotvr1;
                vel1.y = vp1.y + bondDir.y*dotvr1;
                vel1.z = vp1.z + bondDir.z*dotvr1;
                velm[particles.x] = vel1;
            }
            else {
                // Move both particles.

                mixed invTotalMass = RECIP(mass1+mass2);
                mixed dotvr2 = vel2.x*bondDir.x + vel2.y*bondDir.y + vel2.z*bondDir.z;
                mixed4 vb2 = bondDir*dotvr2;
                mixed4 vp2 = vel2-vb2;
                mixed vbCMass = (mass1*dotvr1 + mass2*dotvr2)*invTotalMass;
                dotvr1 -= vbCMass;
                dotvr2 -= vbCMass;
                if (dotvr1 != dotvr2)
                    deltaT = deltaR/fabs(dotvr1-dotvr2);
                if (deltaT > stepSize)
                    deltaT = stepSize;
                mixed vBond = hardwallscaleDrude/SQRT(mass1);
                dotvr1 = -dotvr1*vBond*mass2*invTotalMass/fabs(dotvr1);
                dotvr2 = -dotvr2*vBond*mass1*invTotalMass/fabs(dotvr2);
                mixed dr1 = -deltaR*mass2*invTotalMass + deltaT*dotvr1;
                mixed dr2 = deltaR*mass1*invTotalMass + deltaT*dotvr2;
                dotvr1 += vbCMass;
                dotvr2 += vbCMass;
                pos1.x += bondDir.x*dr1;
                pos1.y += bondDir.y*dr1;
                pos1.z += bondDir.z*dr1;
                pos2.x += bondDir.x*dr2;
                pos2.y += bondDir.y*dr2;
                pos2.z += bondDir.z*dr2;
#ifdef USE_MIXED_PRECISION
                posq[particles.x] = make_real4((real) pos1.x, (real) pos1.y, (real) pos1.z, (real) pos1.w);
                posq[particles.y] = make_real4((real) pos2.x, (real) pos2.y, (real) pos2.z, (real) pos2.w);
                posqCorrection[particles.x] = make_real4(pos1.x-(real) pos1.x, pos1.y-(real) pos1.y, pos1.z-(real) pos1.z, 0);
                posqCorrection[particles.y] = make_real4(pos2.x-(real) pos2.x, pos2.y-(real) pos2.y, pos2.z-(real) pos2.z, 0);
#else
                posq[particles.x] = pos1;
                posq[particles.y] = pos2;
#endif
                vel1.x = vp1.x + bondDir.x*dotvr1;
                vel1.y = vp1.y + bondDir.y*dotvr1;
                vel1.z = vp1.z + bondDir.z*dotvr1;
                vel2.x = vp2.x + bondDir.x*dotvr2;
                vel2.y = vp2.y + bondDir.y*dotvr2;
                vel2.z = vp2.z + bondDir.z*dotvr2;
                velm[particles.x] = vel1;
                velm[particles.y] = vel2;
            }
        }
    }
}
